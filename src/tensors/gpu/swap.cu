#include "cuda_helpers.h"
#include "swap.h"
void copyCpuToGpu(const char * in, char * gpuOut);
void copyGpuToGpu(const char * in, char * gpuOut);

namespace marian {
    namespace swapper {

#ifdef CUDA_FOUND
        void copyCpuToGpu(char * gpuOut, const char * in, size_t count, const marian::DeviceId& deviceId) {
            CUDA_CHECK(hipSetDevice(deviceId.no));
            CUDA_CHECK(hipMemcpy(gpuOut, in, count, hipMemcpyHostToDevice));
        }

        void copyGpuToGpu(char * gpuOut, const char * in, size_t count, const marian::DeviceId& deviceId) {
            CUDA_CHECK(hipSetDevice(deviceId.no));
            CUDA_CHECK(hipMemcpy(gpuOut, in, count, hipMemcpyDeviceToDevice));
        }
#else
        void copyCpuToGpu(char * gpuOut, const char * in, size_t count, const marian::DeviceId& deviceId) {
            ABORT("Copy from CPU to GPU memory is only available with CUDA.");
        }

        void copyGpuToGpu(char * gpuOut, const char * in, size_t count, const marian::DeviceId& deviceId) {
            ABORT("Copy from GPU to GPU memory is only available with CUDA.");
        }
#endif
    }
}
